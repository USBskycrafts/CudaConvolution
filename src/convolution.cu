#include "hip/hip_runtime.h"
#include "convolution.hpp"
#include "race.hpp"
#include <cstdint>
#include <iostream>

#define BLOCK_SIZE 32
#define KERNEL_SIZE 3

static inline void copy_from_image(float *&p_image, Mat &image) {
  // p_image = (float *)malloc(image.rows * image.cols * sizeof(float));
  // for (int i = 0; i < image.rows; i++) {
  //   for (int j = 0; j < image.cols; j++) {
  //     p_image[i * image.cols + j] = image.at<float>(i, j);
  //   }
  // }
  p_image = (float *)image.data;
}

static inline void copy_to_image(Mat &image, float *p_image) {
  // for (int i = 0; i < image.rows; i++) {
  //   for (int j = 0; j < image.cols; j++) {
  //     image.at<float>(i, j) = p_image[i * image.cols + j];
  //   }
  // }
  image.data = (uchar *)p_image;
}

__global__ void convolution_kernel(float *image, uint16_t height,
                                   uint16_t width, float *kernel,
                                   float *o_image) {
  auto index = blockIdx.y * width + blockIdx.x * BLOCK_SIZE + threadIdx.x;
  auto radio = KERNEL_SIZE / 2;
  float sum = 0.0;
  for (int i = 0; i < KERNEL_SIZE; i++) {
    for (int j = 0; j < KERNEL_SIZE; j++) {
      int p = index + (i - radio) * width + (j - radio);
      if (p >= 0 && p < width * height) {
        sum += kernel[i * KERNEL_SIZE + j] * image[p];
      }
    }
  }
  o_image[index] = sum;
}

void convolution(Mat &image, Mat &kernel) {
  float *p_image = nullptr;
  float *p_kernel = nullptr;
  copy_from_image(p_image, image);
  copy_from_image(p_kernel, kernel);
  assert(kernel.cols == kernel.rows && kernel.cols % 2);
  float *k_image = nullptr;
  float *k_kernel = nullptr;
  float *o_image = nullptr;

  // copy to device
  hipMalloc((void **)&k_image, image.cols * image.rows * sizeof(float));
  hipMalloc((void **)&k_kernel, kernel.cols * kernel.rows * sizeof(float));
  hipMalloc((void **)&o_image, image.cols * image.rows * sizeof(float));
  hipMemcpy(k_image, p_image, image.rows * image.cols * sizeof(float),
             hipMemcpyHostToDevice);
  hipMemcpy(k_kernel, p_kernel, kernel.cols * kernel.rows * sizeof(float),
             hipMemcpyHostToDevice);

  // do convolution
  auto grid_size = dim3(image.cols / BLOCK_SIZE + 1, image.rows);
  int start = clock();
  convolution_kernel<<<grid_size, BLOCK_SIZE>>>(k_image, image.rows, image.cols,
                                                k_kernel, o_image);
  int end = clock();
  std::cerr << "running time of convolution is " << (end - start) << "us" << std::endl;
  hipDeviceSynchronize();
  std::cerr << hipGetErrorString(hipGetLastError()) << std::endl;
  // copy to host
  hipMemcpy(p_image, o_image, image.rows * image.cols * sizeof(float),
             hipMemcpyDeviceToHost);
  copy_to_image(image, p_image);
  hipFree(k_image);
  hipFree(k_kernel);
  hipFree(o_image);
}
#undef BLOCK_SIZE